#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include "sssp_common.h"

void matrixInit(float *a, float *b, float *c)
{
    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {   
            a[i * MATRIX_SIZE + j] = rand() / (float)1147654321;
            b[i * MATRIX_SIZE + j] = rand() / (float)1147654321;
            c[i * MATRIX_SIZE + j] = (float)0;
        }
        
    }
}

__device__ void warpReduce(volatile matElement *newSharedB, int threadId)
{
    if(newSharedB[threadId].value > newSharedB[threadId + 32].value){
        newSharedB[threadId].value = newSharedB[threadId + 32].value;
        newSharedB[threadId].row = newSharedB[threadId + 32].row;
        newSharedB[threadId].col = newSharedB[threadId + 32].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 16].value){
        newSharedB[threadId].value = newSharedB[threadId + 16].value;
        newSharedB[threadId].row = newSharedB[threadId + 16].row;
        newSharedB[threadId].col = newSharedB[threadId + 16].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 8].value){
        newSharedB[threadId].value = newSharedB[threadId + 8].value;
        newSharedB[threadId].row = newSharedB[threadId + 8].row;
        newSharedB[threadId].col = newSharedB[threadId + 8].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 4].value){
        newSharedB[threadId].value = newSharedB[threadId + 4].value;
        newSharedB[threadId].row = newSharedB[threadId + 4].row;
        newSharedB[threadId].col = newSharedB[threadId + 4].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 2].value){
        newSharedB[threadId].value = newSharedB[threadId + 2].value;
        newSharedB[threadId].row = newSharedB[threadId + 2].row;
        newSharedB[threadId].col = newSharedB[threadId + 2].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 1].value){
        newSharedB[threadId].value = newSharedB[threadId + 1].value;
        newSharedB[threadId].row = newSharedB[threadId + 1].row;
        newSharedB[threadId].col = newSharedB[threadId + 1].col;
    }
}

__device__ void minBlockReduce(matElement *newSharedB, int threadId)
{
    for (unsigned int stride = (BLOCK_DIM * BLOCK_DIM)/2; stride > 32; stride >>= 1)
    {
        if(threadId < stride)
        {
            if(newSharedB[threadId].value > newSharedB[threadId + stride].value){
                newSharedB[threadId] = newSharedB[threadId + stride];
            }
        }
        __syncthreads();
    }
    if(threadId < 32) warpReduce(newSharedB, threadId);
}

__global__ void find2Min(int16_t firstMinRow, int16_t firstMinCol, float *c, matElement *d_minValueFromEachBlock)
{
    int16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    int16_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int16_t threadId = threadIdx.y * BLOCK_DIM + threadIdx.x;

    __shared__ matElement sharedC[BLOCK_DIM * BLOCK_DIM];

    if(row == 0 && col == 0) c[firstMinRow * MATRIX_SIZE + firstMinCol] = __FLT_MAX__;
    
    __syncthreads();

    sharedC[threadId].value = c[row * MATRIX_SIZE + col];
    sharedC[threadId].row = row;
    sharedC[threadId].col = col;
    __syncthreads();

    minBlockReduce(sharedC, threadId);
    if(threadId == 0){   
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].value = sharedC[0].value;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].row = sharedC[0].row;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].col = sharedC[0].col;
    }

    // if(row == 0 && col ==0) c[firstMinRow * MATRIX_SIZE + firstMinCol] = tempVal;                   //replace the first min val with the original since we replaced it with FLT_MAX for finding second min
}

__global__ void tiledMatrixMultiply(float *a, float *b, float *c, matElement *d_minValueFromEachBlock)
{
    int16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    int16_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int16_t threadId = threadIdx.y * BLOCK_DIM + threadIdx.x;

    __shared__ float sharedA[BLOCK_DIM * BLOCK_DIM];
    __shared__ float sharedB[BLOCK_DIM * BLOCK_DIM * sizeof(matElement)];                    

    float temp = 0;

    for (int i = 0; i < MATRIX_SIZE / TILE_SZE; i++)
    {
        sharedA[threadId] = a[row * MATRIX_SIZE + (i * TILE_SZE + threadIdx.x)];                 //index into the global a with the global row (since we are tiling across x dimention of a) and each thread's tile 
        sharedB[threadId] = b[(i * TILE_SZE + threadIdx.y) * MATRIX_SIZE + col];                 //index into the global b with each thread's tile idexes (since we are tiling across y dimention of b) and globale column 
        __syncthreads();                                                                         //make sure all values of the sub-matrices are loaded by thre threads before proceding

        for (int j = 0; j < TILE_SZE; j++)
        {
            temp += sharedA[threadIdx.y * TILE_SZE + j] * sharedB[j * TILE_SZE + threadIdx.x];
        }

        __syncthreads();                                                                         //make sure all sub-matrix calculation is done by threads before advancing to the next sub-matricies

    }
    matElement *newSharedB = (matElement*) sharedB;                                              //reuse shared mem for finding min element

    newSharedB[threadId].value = temp;
    newSharedB[threadId].row = row;
    newSharedB[threadId].col = col;
    __syncthreads();
    
    c[row * MATRIX_SIZE + col] = temp;

    minBlockReduce(newSharedB, threadId);
    if(threadId == 0){   
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].value = newSharedB[0].value;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].row = newSharedB[0].row;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].col = newSharedB[0].col;
    }
}

extern float* computeMatrixMult(matElement *minElement)
{
    struct timeval start_time, end_time;
    double exec_time;
    minElement[0].value = __FLT_MAX__;
    minElement[1].value = __FLT_MAX__;

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    matElement *h_minValueFromEachBlock;
    matElement *d_minValueFromEachBlock;

    size_t size = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);

    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    h_minValueFromEachBlock = (matElement*)malloc((MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement));

    CHECK(hipHostMalloc(&d_a, size));
    CHECK(hipHostMalloc(&d_b, size));
    CHECK(hipHostMalloc(&d_c, size));
    CHECK(hipHostMalloc(&d_minValueFromEachBlock, (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement)));

    matrixInit(h_a, h_b, h_c);

    dim3 blockPerGrid(MATRIX_SIZE / BLOCK_DIM , MATRIX_SIZE / BLOCK_DIM);
    dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);

    
    gettimeofday(&start_time, NULL);

    CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));
    

    tiledMatrixMultiply<<<blockPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, d_minValueFromEachBlock);

    CHECK(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_minValueFromEachBlock, d_minValueFromEachBlock, (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement), hipMemcpyDeviceToHost));
    for (int i = 0; i < (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM); i++)
    {
        if(h_minValueFromEachBlock[i].value < minElement[0].value)
        {
            minElement[0].value = h_minValueFromEachBlock[i].value;           
            minElement[0].row = h_minValueFromEachBlock[i].row;
            minElement[0].col = h_minValueFromEachBlock[i].col;
        }
    }

    find2Min<<<blockPerGrid, threadsPerBlock>>>(minElement[0].row, minElement[0].col, d_c, d_minValueFromEachBlock);

    CHECK(hipMemcpy(h_minValueFromEachBlock, d_minValueFromEachBlock, (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement), hipMemcpyDeviceToHost));
    
    d_c[minElement[0].row * MATRIX_SIZE + minElement[0].col] = minElement[0].value;
    
    for (int i = 0; i < (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM); i++)
    {
        if(h_minValueFromEachBlock[i].value < minElement[1].value)
        {
            minElement[1].value = h_minValueFromEachBlock[i].value;           
            minElement[1].row = h_minValueFromEachBlock[i].row;
            minElement[1].col = h_minValueFromEachBlock[i].col;
        }
    }
    gettimeofday(&end_time, NULL);

    free(h_a);
    free(h_b);

    hipFree(d_a);
    hipFree(d_b);

    exec_time = (double)(end_time.tv_sec - start_time.tv_sec) + (double)(end_time.tv_usec - start_time.tv_usec)/(double)1000000;

    std::cout<<"Execution time - "<<exec_time<<std::endl;
    
    std::cout<<"Matrix size - "<<MATRIX_SIZE<<std::endl;

    std::cout<<"Min value 1 (val, row, col) - ("<<minElement[0].value<<", "<<minElement[0].row<<", "<<minElement[0].col<<")"<<std::endl;

    std::cout<<"Min value 2 (val, row, col) - ("<<minElement[1].value<<", "<<minElement[1].row<<", "<<minElement[1].col<<")"<<std::endl;

    return d_c;

}