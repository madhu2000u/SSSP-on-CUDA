#include "hip/hip_runtime.h"
#include <iostream>
#include "sssp_common.h"




float* computeMatrixMult(matElement*);

void setUpArrays(float *d_c, int *vertex, int *edges, bool *threadMask, float* cost, float* intermediateCost, int* path, matElement* minElement)
{   
    int edgeIndex = 0;
    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {   
            threadMask[i * MATRIX_SIZE + j] = false;
            cost[i * MATRIX_SIZE + j] = __FLT_MAX__;
            intermediateCost[i * MATRIX_SIZE + j] = __FLT_MAX__;
            path[i * MATRIX_SIZE + j] = -1;

            vertex[i * MATRIX_SIZE + j] = edgeIndex;
            if((j + 1) < MATRIX_SIZE) edges[edgeIndex++] = i * MATRIX_SIZE + (j + 1);
        
            if((i + 1) < MATRIX_SIZE) edges[edgeIndex++] = (i + 1) * MATRIX_SIZE + j;

            if((j - 1) >= 0) edges[edgeIndex++] = i * MATRIX_SIZE + (j - 1);

            if((i - 1) >= 0) edges[edgeIndex++] = (i - 1) * MATRIX_SIZE + j;

        }

        threadMask[minElement[0].row * MATRIX_SIZE + minElement[0].col] = true;             //Make the thread of source vertex executable initially since that is the starting point
        cost[minElement[0].row * MATRIX_SIZE + minElement[0].col] = 0.0f;                   //Cost from source to source is 0
        intermediateCost[minElement[0].row * MATRIX_SIZE + minElement[0].col] = 0.0f;       
        
    }
    
}

void printNeighbors(int index, float *d_c, int *vertex, int* edges)
{
    for (int i = vertex[index]; i < vertex[index + 1]; i++)
    {
        std::cout<<i<<std::endl;
        std::cout<<d_c[edges[i]]<<std::endl;
        std::cout<<"\n";
    }
    
}

void printPath(int *path)
{
    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {
            printf("(%d, %d),  ", i * MATRIX_SIZE + j, path[i * MATRIX_SIZE + j]);
        }
        
        
    }
}

typedef struct pathElement
{
    float value;
    int pathIndex;
};


unsigned long long int __pathElement_as_ulli(pathElement *pathElement)
{
    unsigned long long int ulli = (unsigned long long int) pathElement;
    return ulli;

}

__device__ __forceinline__ float atomicMinPath(float *intermediateAddr, int *pathAddr, float value, int indexOfCallingThreadToPointToForCorrespondingAddrToGivePath)
{
    int currentIntermediateCost = __float_as_int(*intermediateAddr);                                            //reinterpret to int since atomicCAS() requires int
    int currentPath = *pathAddr;
    while (value < __int_as_float(currentIntermediateCost))                                         
    {//atomicCAS_block()
        int old = currentPath;
        currentPath = atomicCAS(pathAddr, old, indexOfCallingThreadToPointToForCorrespondingAddrToGivePath);                //if *addr == old then it puts value into addr and returns old else it does nothing and just retunrs whatever was there in addr
        if(currentPath == old) break;                                                   //if value was successfully put into addr then the current thread was successful in it's atomic operation else it has to re-run with the new "current value" from addr(that might have been changed by another thread's atomic operation) and do the swapping again
    }
    return currentPath;
    
}

__device__ __forceinline__ float atomicMin(float *addr, float value)
{
    int current = __float_as_int(*addr);                                            //reinterpret to int since atomicCAS() requires int
    while (value < __int_as_float(current))                                         
    {
        int old = current;
        current = atomicCAS((int*)addr, old, __float_as_int(value));                //if *addr == old then it puts value into addr and returns old else it does nothing and just retunrs whatever was there in addr
        if(current == old) break;                                                   //if value was successfully put into addr then the current thread was successful in it's atomic operation else it has to re-run with the new "current value" from addr(that might have been changed by another thread's atomic operation) and do the swapping again
    }
    return __int_as_float(current);
    
}

__global__ void computePaths(float *d_c, int *vertex, int *edges, bool *threadMask, float *cost, float *intermediateCost, int* path)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int threadIndex = row * MATRIX_SIZE + col;                              //this is the global thread index to index into the entire matrix and not just for threads within each block
    if(threadMask[threadIndex])
    {
        //threadMask[threadIndex] = false;
        for (int i = vertex[threadIndex]; i < vertex[threadIndex + 1]; i++)
        {   
            atomicMinPath(&intermediateCost[edges[i]], &path[edges[i]], (cost[threadIndex] + d_c[edges[i]]), threadIndex);
        }
        
    }
}

__global__ void computeIntermediates(float *d_c, int *vertex, int *edges, bool *threadMask, float *cost, float *intermediateCost)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int threadIndex = row * MATRIX_SIZE + col;                              //this is the global thread index to index into the entire matrix and not just for threads within each block
    if(threadMask[threadIndex])
    {
        threadMask[threadIndex] = false;
        for (int i = vertex[threadIndex]; i < vertex[threadIndex + 1]; i++)
        {   
            atomicMin(&intermediateCost[edges[i]], (cost[threadIndex] + d_c[edges[i]]));
        }
        
    }
}

__global__ void computeFinalCostsAndPath(bool *d_done, int *vertex, int *edges, bool *threadMask, float *cost, float *intermediateCost)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int threadIndex = row * MATRIX_SIZE + col;
    if(intermediateCost[threadIndex] < cost[threadIndex])
    {
        cost[threadIndex] = intermediateCost[threadIndex];
        threadMask[threadIndex] = true;                                                 //since cost of this vertex changed, make it executable again to update it's neigbours
        *d_done = false;                                                                 //no atomicity required as all threads write false value only
    }

    intermediateCost[threadIndex] = cost[threadIndex];

}

int main()
{

    int *vertex, *edges, *path;
    float *d_c;
    float *cost, *intermediateCost;
    bool *threadMask;

    bool h_done = false;
    bool *d_done_ptr;

    matElement minElement[2];
    size_t size = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);
    //Note: Diagnonal neighbours are not considered
    int numEdges = (4 * 2                                       /*Each corner values in matrix has 2 neighbours*/ 
                    + ((MATRIX_SIZE - 2) * 3) * 4               /*Each element of the 4 boundary sides excluding the 2 corner elements for each boundary side has 3 neighbours*/ 
                    + (MATRIX_SIZE - 2) * 4 * (MATRIX_SIZE - 2) /*Each element not on the boundary has 4 neighbours*/);
    
    //Compute and get the pointer to the result matrix of the matrix muliplications
    d_c = computeMatrixMult(minElement);

    //Use test data for sssp checking
    #if(TEST)
    float test_data[16] = {1.2, 5.4, 1.0f, 1.0f, 9.7, 4.9, 1.0f, 7.6, 6.1, 8.4, 1.0f, 11.5, 14.3, 15.8, 1.0f, 17.7};
    CHECK(hipMemcpy(d_c, &test_data, size, hipMemcpyHostToDevice));

    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {
            std::cout<<d_c[i * MATRIX_SIZE + j]<<"  ";
        }
        printf("\n");
        
    }
    //Test values for source and target
    minElement[0].row = 0; minElement[0].col = 3;
    minElement[1].row = 3; minElement[1].col = 3;
    #endif

    // vertex = (int*)malloc(((MATRIX_SIZE * MATRIX_SIZE) + 1) * sizeof(int));                            
    // threadMask = new bool[MATRIX_SIZE * MATRIX_SIZE];
    // edges = (int*)malloc( numEdges * sizeof(int));

    // cost = (float*)malloc(size);
    // intermediateCost = (float*)malloc(size);                                                          

   

    

    //Setup CUDA device memories for the data
    CHECK(hipHostMalloc(&vertex, ((MATRIX_SIZE * MATRIX_SIZE) + 1) * sizeof(int)));                     // + 1 because we need a location at the end of the vertex that stores the ending index of the edge
    CHECK(hipHostMalloc(&edges, numEdges * sizeof(int)));
    CHECK(hipHostMalloc(&threadMask, MATRIX_SIZE * MATRIX_SIZE * sizeof(bool)));
    CHECK(hipHostMalloc(&cost, size));
    CHECK(hipHostMalloc(&intermediateCost, numEdges * sizeof(float)));                                  //each neighbor need not have it's own cost location because the intermediate cost for a vertex is the same memory location updated by all neighbouring threads.
    CHECK(hipHostMalloc(&path, size));
    CHECK(hipHostMalloc(&d_done_ptr, sizeof(bool)));

    setUpArrays(d_c, vertex, edges, threadMask, cost, intermediateCost, path, minElement);
    vertex[MATRIX_SIZE * MATRIX_SIZE] = numEdges;                                                     //last value in vertex is total numEdges so that we can use the starting and ending index when getting the neighbors


    dim3 blockPerGrid(MATRIX_SIZE / BLOCK_DIM , MATRIX_SIZE / BLOCK_DIM);
    dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);

    //Start computing SSSP
    while(!h_done)
    {
        h_done = true;
        //memcpy h_done to d_done
        CHECK(hipMemcpy(d_done_ptr, &h_done, sizeof(bool), hipMemcpyHostToDevice));

        computePaths<<<blockPerGrid, threadsPerBlock>>>(d_c, vertex, edges, threadMask, cost, intermediateCost, path);
        hipDeviceSynchronize();

        //call kernel 1
        computeIntermediates<<<blockPerGrid, threadsPerBlock>>>(d_c, vertex, edges, threadMask, cost, intermediateCost);
        hipDeviceSynchronize();

        //call kernel 2
        computeFinalCostsAndPath<<<blockPerGrid, threadsPerBlock>>>(d_done_ptr, vertex, edges, threadMask, cost, intermediateCost);
        hipDeviceSynchronize();

        //memcpy d_done to h_done
        CHECK(hipMemcpy(&h_done, d_done_ptr, sizeof(bool), hipMemcpyDeviceToHost));
        
        printPath(path);
        printf("\n");

    }
    
    
    printf("\ncost of target - %f\n", d_c[minElement[0].row * MATRIX_SIZE + minElement[0].col] + cost[minElement[1].row * MATRIX_SIZE + minElement[1].col] - d_c[minElement[1].row * MATRIX_SIZE + minElement[1].col]);       //include source's weight and exclude target's weight

}